#include "hip/hip_runtime.h"
/**************************************************************************
** Copyright (C) 2010-2017 Toshinobu Hondo, Ph.D.
** Copyright (C) 2013-2017 MS-Cheminformatics LLC, Toin, Mie Japan
*
** Contact: toshi.hondo@qtplatz.com
**
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

* Redistributions of source code must retain the above copyright notice, this
  list of conditions and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright notice,
  this list of conditions and the following disclaimer in the documentation
  and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names of its
  contributors may be used to endorse or promote products derived from
  this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
**************************************************************************/

#include "aftypes.hpp"
#include <stdio.h>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
#include <arrayfire.h>
#include <af/hip/hip_runtime.h>

typedef uint8_t rgb_type;

namespace af_extension {

    enum RGB { Red = 0, Green = 1, Blue = 2 };
    
    template<typename T>
    struct ColorTable {
        const int num_;
        const T * rgb_;

        __device__ ColorTable( int num, const T* rgb ) : num_( num ), rgb_(rgb) {
        }

        __device__ inline T color( int level, RGB code ) const {
            return rgb_[ level + num_ * code ];
        }

        __device__ T operator ()( int level, float frac, RGB code ) const {
            if ( level == 0 )
                return color( level, code );
            else if ( level == num_ )
                return color( num_ - 1, code );
            else if ( level > 0 )
                return ( color( level, code ) - color ( level - 1, code ) ) * frac + color( level - 1, code );
            else
                return T(0);
        }        
    };
}

__global__
void
colormap_kernel( const int num, const float * d_x, rgb_type * d_y
                 , const int nlevels, const float * d_levels, const float * d_colors )
{
    const int id = blockIdx.x * blockDim.x + threadIdx.x;

    float r(0), g(0), b(0), frac(0);
    int level = 0;

    af_extension::ColorTable<float> table( nlevels, d_colors );

    if ( id < num ) {
        while ( level < nlevels ) {
            if ( d_x[ id ] < d_levels[ level ] )
                break;
            ++level;
        }
        if ( level > 0 ) {
            frac = ( d_x[ id ] - d_levels[ level - 1 ] ) / ( d_levels[ level ] - d_levels[ level - 1 ] );
            r = table( level, frac, af_extension::Red  );
            g = table( level, frac, af_extension::Green );
            b = table( level, frac, af_extension::Blue );
        }

        d_y[id + num * 0] = r * 255;
        d_y[id + num * 1] = g * 255;
        d_y[id + num * 2] = b * 255;
    }
}

af::array
colorMap( const af::array& gray, const af::array& levels, const af::array& colors )
{
    gray.eval(); // Ensure any JIT kernels have executed
    levels.eval();
    colors.eval();

    int cuda_id = afcu::getNativeId( af::getDevice() ); // Determine ArrayFire's CUDA stream
    hipStream_t af_cuda_stream = afcu::getStream( cuda_id );

    const int num = gray.dims(0) * gray.dims(1);

    const float * d_gray = gray.device< float >();

    using namespace arrayfire;
    
    // result array

    af::array rgb = af::constant< rgb_type >( 0, gray.dims(0), gray.dims(1), 3, af_type_value< rgb_type >::value );
    rgb_type * d_rgb = rgb.device< rgb_type >();

    const float * d_levels = levels.device< float >();
    const float * d_colors = colors.device< float >();

    const int threads = 256;
    const int blocks = (num / threads) + ((num % threads) ? 1 : 0 );

    colormap_kernel <<< blocks, threads, 0, af_cuda_stream >>> ( num, d_gray, d_rgb, levels.dims(0), d_levels, d_colors );

    hipDeviceSynchronize();

    rgb.unlock();

    return rgb;
}

